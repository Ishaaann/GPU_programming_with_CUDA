#include "hip/hip_runtime.h"
#include "../include/utils.h"
#include <hip/hip_runtime.h>

// Naive GPU matrix multiplication kernel
__global__ void naive_matrix_mul_kernel(const float* A, const float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

void gpu_naive_matrix_multiply(const float* A, const float* B, float* C, int N) {
    // Device memory pointers
    float *d_A, *d_B, *d_C;
    
    // Calculate memory size
    size_t size = N * N * sizeof(float);
    
    // Allocate device memory
    CUDA_CHECK(hipMalloc(&d_A, size));
    CUDA_CHECK(hipMalloc(&d_B, size));
    CUDA_CHECK(hipMalloc(&d_C, size));
    
    // Copy input matrices to device
    CUDA_CHECK(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice));
    
    // Define block and grid dimensions
    const int BLOCK_SIZE = 16;
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);
    
    // Launch kernel
    naive_matrix_mul_kernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, N);
    
    // Check for kernel launch errors
    CUDA_CHECK(hipGetLastError());
    
    // Synchronize to ensure kernel completion
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Copy result back to host
    CUDA_CHECK(hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost));
    
    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
