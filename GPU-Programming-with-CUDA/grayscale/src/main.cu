#include <iostream>
#include <string>
#include <vector>
#include <dirent.h>
#include <sys/stat.h> // Required for directory creation

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#include "grayscale.cuh"

// Helper function to check for CUDA errors
void checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(result));
        exit(1);
    }
}

// Function to create a directory if it doesn't exist
void createDirectory(const std::string& path) {
    struct stat info;
    if (stat(path.c_str(), &info) != 0) {
        mkdir(path.c_str(), 0755);
    } else if (!(info.st_mode & S_IFDIR)) {
        fprintf(stderr, "Error: %s is not a directory!\n", path.c_str());
        exit(1);
    }
}


int main() {
    const std::string input_dir = "input_data";
    const std::string output_dir = "output_data";

    // Create output directory if it doesn't exist
    createDirectory(output_dir);

    DIR *dir;
    struct dirent *ent;

    if ((dir = opendir(input_dir.c_str())) != NULL) {
        std::cout << "Starting batch image processing..." << std::endl;

        // Iterate over all the files in the input directory
        while ((ent = readdir(dir)) != NULL) {
            std::string filename = ent->d_name;
            if (filename == "." || filename == "..") {
                continue;
            }

            std::string input_path = input_dir + "/" + filename;

            // 1. Load image from disk using stb_image
            int width, height, channels;
            unsigned char *h_input_img = stbi_load(input_path.c_str(), &width, &height, &channels, 0);

            if (h_input_img == NULL) {
                std::cerr << "Error loading image: " << input_path << std::endl;
                continue;
            }

            // We only process RGB or RGBA images
            if (channels < 3) {
                std::cout << "Skipping non-color image: " << filename << std::endl;
                stbi_image_free(h_input_img);
                continue;
            }

            std::cout << "Processing: " << filename << " (" << width << "x" << height << ")" << std::endl;

            // 2. Allocate memory on the host and device
            size_t img_size = width * height * channels * sizeof(unsigned char);
            size_t gray_img_size = width * height * sizeof(unsigned char);

            unsigned char *h_output_gray_img = (unsigned char*)malloc(gray_img_size);
            unsigned char *d_input_img, *d_output_gray_img;

            checkCuda(hipMalloc((void**)&d_input_img, img_size));
            checkCuda(hipMalloc((void**)&d_output_gray_img, gray_img_size));

            // 3. Copy image data from host to device
            checkCuda(hipMemcpy(d_input_img, h_input_img, img_size, hipMemcpyHostToDevice));

            // 4. Launch the CUDA kernel
            convertToGrayscale(d_input_img, d_output_gray_img, width, height, channels);

            // Check for any errors during kernel execution
            checkCuda(hipGetLastError());
            checkCuda(hipDeviceSynchronize());

            // 5. Copy the result back from device to host
            checkCuda(hipMemcpy(h_output_gray_img, d_output_gray_img, gray_img_size, hipMemcpyDeviceToHost));

            // 6. Save the grayscale image to disk
            std::string output_path = output_dir + "/gray_" + filename;
            // Save as PNG to avoid compression artifacts and handle single channel easily
            size_t dot_pos = output_path.find_last_of(".");
            if (dot_pos != std::string::npos) {
                 output_path.replace(dot_pos, output_path.length() - dot_pos, ".png");
            } else {
                 output_path += ".png";
            }
            stbi_write_png(output_path.c_str(), width, height, 1, h_output_gray_img, width * sizeof(unsigned char));

            // 7. Free memory
            stbi_image_free(h_input_img);
            free(h_output_gray_img);
            hipFree(d_input_img);
            hipFree(d_output_gray_img);
        }
        closedir(dir);
        std::cout << "Processing complete. Results are in the 'output_data' directory." << std::endl;
    } else {
        std::cerr << "Error: Could not open input directory '" << input_dir << "'. Please create it and add images." << std::endl;
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}
